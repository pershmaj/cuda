
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>

#include "/usr/local/cuda/include/cuda_runtime.h"

#define CUDA_CHECK_RETURN( value ) {                            \
    hipError_t err = value;                                    \
    if(err != hipSuccess) {                                    \
        fprintf(stderr, "Error %s at line %d in file %s \n",    \
            hipGetErrorString(err), __LINE__, __FILE__);       \
        exit(1);                                                \
    }                                                           \
}

using namespace std;

#define VECT_SIZE (2u)
#define BLOCK_SIZE (2u)

__global__ void multiplyMatrixes(int *data1, int *data2, int *data3) {
    int row = threadIdx.x + blockIdx.x * blockDim.x;
    int col = threadIdx.y + blockIdx.y * blockDim.y;
    if(row < VECT_SIZE && col < VECT_SIZE) {
        int r = 0;
        for(int i = 0; i < VECT_SIZE; i++) {
            r+= data1[row * VECT_SIZE + i] * data2[col + VECT_SIZE * i]; 
        }
        data3[row * VECT_SIZE + col] = r;
    }
}

int main(){
    const int MatrixSize = VECT_SIZE * VECT_SIZE * sizeof(int);

    int *h_data1 = (int*) malloc(MatrixSize);
    int *h_data2 = (int*) malloc(MatrixSize);
    int *h_data3 = (int*) malloc(MatrixSize);
    int *d_data1 = NULL;
    int *d_data2 = NULL;
    int *d_data3 = NULL;

    CUDA_CHECK_RETURN(hipMalloc(&d_data1, MatrixSize));
    CUDA_CHECK_RETURN(hipMalloc(&d_data2, MatrixSize));
    CUDA_CHECK_RETURN(hipMalloc(&d_data3, MatrixSize));
    //kernel config
    for (int i = 0; i < VECT_SIZE * VECT_SIZE; i++) {
        h_data1[i] = 3;//i+1;
        h_data2[i] = 3;//i+1;
    }

    CUDA_CHECK_RETURN( hipMemcpy( d_data1, h_data1, MatrixSize, hipMemcpyHostToDevice ) );
    CUDA_CHECK_RETURN( hipMemcpy( d_data2, h_data2, MatrixSize, hipMemcpyHostToDevice ) );

    // dim2 blockSize( BLOCK_SIZE, BLOCK_SIZE );
	// dim2 gridSize( (VECT_SIZE + BLOCK_SIZE - 1) / BLOCK_SIZE,
	// 			   (VECT_SIZE + BLOCK_SIZE - 1) / BLOCK_SIZE );
    // int blockSize = BLOCK_SIZE;
    // int gridSize = (VECT_SIZE + BLOCK_SIZE - 1) / BLOCK_SIZE;

    dim3 gridSize(ceilf(VECT_SIZE/(float)BLOCK_SIZE), ceilf(VECT_SIZE/(float)BLOCK_SIZE), 1);
    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE, 1);
    
    //kernel execution
    multiplyMatrixes<<<gridSize, blockSize>>>(d_data1, d_data2, d_data3);
    //await for kernel computation
    CUDA_CHECK_RETURN(hipDeviceSynchronize());
    // copy data (works both ways)
    CUDA_CHECK_RETURN(hipMemcpy(h_data3, d_data3, MatrixSize, hipMemcpyDeviceToHost));

    for(int i = 0; i < VECT_SIZE * VECT_SIZE; i++) {
        cout << h_data3[i];
        cout << ", ";
    }

    free(h_data1);
    free(h_data2);
    free(h_data3);
    CUDA_CHECK_RETURN(hipFree(d_data1));
    CUDA_CHECK_RETURN(hipFree(d_data2));
    CUDA_CHECK_RETURN(hipFree(d_data3));
}