#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <chrono>

#include "/usr/local/cuda/include/hip/hip_runtime.h"

#define CUDA_CHECK_RETURN( value ) {                            \
    hipError_t err = value;                                    \
    if(err != hipSuccess) {                                    \
        fprintf(stderr, "Error %s at line %d in file %s \n",    \
            hipGetErrorString(err), __LINE__, __FILE__);       \
        exit(1);                                                \
    }                                                           \
}

using namespace std;

#define VECT_SIZE (10u)
#define BLOC_SIZE (128u)

__global__ void sumVector(int *data1, int *data2, int *data3) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if(i < VECT_SIZE) {
        data3[i] = data1[i] + data2[i];
    }

}

int main(){
    int h_matrix1[VECT_SIZE][VECT_SIZE];
    int h_matrix2[VECT_SIZE][VECT_SIZE];
    int **d_data1 = NULL;
    int **d_data2 = NULL;
    int **d_data3 = NULL;


    CUDA_CHECK_RETURN(hipMalloc(&d_data1, VECT_SIZE * sizeof(int)));
    CUDA_CHECK_RETURN(hipMalloc(&d_data2, VECT_SIZE * sizeof(int)));
    CUDA_CHECK_RETURN(hipMalloc(&d_data3, VECT_SIZE * sizeof(int)));
    //kernel config

     for(int i = 0; i < VECT_SIZE; i++) {
        CUDA_CHECK_RETURN(hipMalloc(&d_data1[i], VECT_SIZE * sizeof(int)));
        CUDA_CHECK_RETURN(hipMalloc(&d_data2[i], VECT_SIZE * sizeof(int)));
        CUDA_CHECK_RETURN(hipMalloc(&d_data3[i], VECT_SIZE * sizeof(int)));
    }

    for(int i = 0; i < VECT_SIZE * VECT_SIZE; i++) {
        *h_matrix1[i] = 2;
        *h_matrix2[i] = 2;
    }

    int blockSize = BLOC_SIZE;
    int gridSize = (VECT_SIZE + BLOC_SIZE - 1) / BLOC_SIZE;
    
    //kernel execution
    fillVector<<<gridSize, blockSize>>>(d_data1);
    //await for kernel computation
    CUDA_CHECK_RETURN(hipDeviceSynchronize());//kernel execution
    fillVector<<<gridSize, blockSize>>>(d_data2);
    //await for kernel computation
    CUDA_CHECK_RETURN(hipDeviceSynchronize());
    //kernel execution
    sumVector<<<gridSize, blockSize>>>(d_data1, d_data2, d_data3);
    //await for kernel computation
    CUDA_CHECK_RETURN(hipDeviceSynchronize());
    // copy data (works both ways)
    CUDA_CHECK_RETURN(hipMemcpy(h_data, d_data3, VECT_SIZE * sizeof(int), hipMemcpyDeviceToHost));

    for(int i = 0; i < VECT_SIZE * VECT_SIZE; i++) {
        cout << h_matrix1[i];
        if( i < VECT_SIZE) {
            cout << ", ";
        }
    }

    free(h_data);
    CUDA_CHECK_RETURN(hipFree(d_data1));
    CUDA_CHECK_RETURN(hipFree(d_data2));
    CUDA_CHECK_RETURN(hipFree(d_data3));
}