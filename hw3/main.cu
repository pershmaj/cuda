
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <chrono>
#include "/usr/local/cuda/include/cuda_runtime.h"

#define CUDA_CHECK_RETURN( value ) {                            \
    hipError_t err = value;                                    \
    if(err != hipSuccess) {                                    \
        fprintf(stderr, "Error %s at line %d in file %s \n",    \
            hipGetErrorString(err), __LINE__, __FILE__);       \
        exit(1);                                                \
    }                                                           \
}

using namespace std;

#define VECT_SIZE (1048576u)
#define BLOCK_SIZE (16u)

// 32u -> 25 763ns
// 512u -> 278 967ns
// 1048576u -> 5369ns
// 134217728u -> 83ns
// 1073741824u -> 140ns


__global__ void multiplyMatrixes(int *A, int *B, int *C) {
    // int row = threadIdx.x + blockIdx.x * blockDim.x;
    // int col = threadIdx.y + blockIdx.y * blockDim.y;
    // if(row < VECT_SIZE && col < VECT_SIZE) {
    //     int r = 0;
    //     for(int i = 0; i < VECT_SIZE; i++) {
    //         r+= data1[row * VECT_SIZE + i] * data2[col + VECT_SIZE * i]; 
    //     }
    //     data3[row * VECT_SIZE + col] = r;
    // }

    int blockRow = blockIdx.x;
    int blockCol = blockIdx.y;
    int row = threadIdx.y;
    int col = threadIdx.x;
    int Cvalue = 0;


    int* Csub = &C[VECT_SIZE * BLOCK_SIZE * blockRow + BLOCK_SIZE * blockCol];

    for(int m = 0; m < VECT_SIZE / BLOCK_SIZE; m++) {
        const int* Asub = &A[VECT_SIZE * BLOCK_SIZE * blockRow + BLOCK_SIZE * m];
        const int* Bsub = &B[VECT_SIZE * BLOCK_SIZE * m + BLOCK_SIZE * blockCol];

        __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
        __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

        As[row][col] = Asub[row * VECT_SIZE + col];
        Bs[row][col] = Bsub[row * VECT_SIZE + col];

        __syncthreads();

        for(int e = 0; e < BLOCK_SIZE; e++) {
            Cvalue += As[row][e] * Bs[e][row];
        }

        __syncthreads();
    }

    Csub[row * VECT_SIZE + col] = Cvalue;
}

int main(){
    const long MatrixSize = VECT_SIZE * VECT_SIZE * sizeof(int);

    int *h_data1 = (int*) malloc(MatrixSize);
    int *h_data2 = (int*) malloc(MatrixSize);
    int *h_data3 = (int*) malloc(MatrixSize);
    int *d_data1 = NULL;
    int *d_data2 = NULL;
    int *d_data3 = NULL;

    CUDA_CHECK_RETURN(hipMalloc(&d_data1, MatrixSize));
    CUDA_CHECK_RETURN(hipMalloc(&d_data2, MatrixSize));
    CUDA_CHECK_RETURN(hipMalloc(&d_data3, MatrixSize));
    //kernel config
    for (int i = 0; i < VECT_SIZE * VECT_SIZE; i++) {
        h_data1[i] = random();//i+1;
        h_data2[i] = random();//i+1;
    }

    CUDA_CHECK_RETURN( hipMemcpy( d_data1, h_data1, MatrixSize, hipMemcpyHostToDevice ) );
    CUDA_CHECK_RETURN( hipMemcpy( d_data2, h_data2, MatrixSize, hipMemcpyHostToDevice ) );

    dim3 gridSize(ceilf(VECT_SIZE/(float)BLOCK_SIZE), ceilf(VECT_SIZE/(float)BLOCK_SIZE), 1);
    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE, 1);
    

    std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();
    
    //kernel execution
    multiplyMatrixes<<<gridSize, blockSize>>>(d_data1, d_data2, d_data3);
    //await for kernel computation
    CUDA_CHECK_RETURN(hipDeviceSynchronize());

    std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();

    std::cout << "Time difference = " << std::chrono::duration_cast<std::chrono::microseconds>(end - begin).count() << "[µs]" << std::endl;
    std::cout << "Time difference = " << std::chrono::duration_cast<std::chrono::nanoseconds> (end - begin).count() << "[ns]" << std::endl;
    // copy data (works both ways)
    CUDA_CHECK_RETURN(hipMemcpy(h_data3, d_data3, MatrixSize, hipMemcpyDeviceToHost));

    // for(int i = 0; i < VECT_SIZE * VECT_SIZE; i++) {
        // cout << h_data3[i];
        // cout << "\t";
        // if(i % VECT_SIZE - 1 == 0) cout << endl;
    // }

    free(h_data1);
    free(h_data2);
    free(h_data3);
    CUDA_CHECK_RETURN(hipFree(d_data1));
    CUDA_CHECK_RETURN(hipFree(d_data2));
    CUDA_CHECK_RETURN(hipFree(d_data3));
}